#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <GL/glut.h>

#define M_PI 3.141592653589793238462643

int timeGetTime(){return 0;};

#define IN_CUDA_ENV
//#ifdef _WIN32
//#  define WINDOWS_LEAN_AND_MEAN
//#  include <windows.h>
//#endif
#include <cutil.h>
#include <cuda_gl_interop.h>
#include "cutil_math.h"
////////////////////////////////////////////////////////////////////////////////
#define C_CHECK_GL_ERROR() ChkGLError(__FILE__, __LINE__)
////////////////////////////////////////////////////////////////////////////////
extern "C" bool pboRegister(int pbo);
extern "C" void pboUnregister(int pbo);
extern "C" void* cuda_main( int pbo_out, 
						   int size_x, 
						   int size_y, 
						   int size_z, 
						   int level, 
						   float box_x,float box_y,float box_z,
						   float pos_x,float pos_y,float pos_z );
////////////////////////////////////////////////////////////////////////////////
#define THREAD_COUNT_X 16
#define THREAD_COUNT_Y 16
#define THREAD_COUNT_Z 1
#define uchar unsigned char  
////////////////////////////////////////////////////////////////////////////////
void gpu_memcpy(void* dst, void* src, int size)
{
	CUDA_SAFE_CALL( hipMemcpy( dst, src, size, hipMemcpyHostToDevice) );
	CUT_CHECK_ERROR("hipMemcpy hipMemcpyHostToDevice failed");
}
////////////////////////////////////////////////////////////////////////////////
void cpu_memcpy(void* dst, void* src, int size)
{
	CUDA_SAFE_CALL( hipMemcpy( dst, src, size, hipMemcpyDeviceToHost) );
	CUT_CHECK_ERROR("hipMemcpy hipMemcpyDeviceToHost failed");
}
////////////////////////////////////////////////////////////////////////////////
void* gpu_malloc(int size)
{
	void* ptr=0;	
	CUDA_SAFE_CALL( hipMalloc( (void**) &ptr, size ) );
	CUT_CHECK_ERROR("hipMalloc failed");
	if(ptr==0){printf("\ncudaMalloc %d MB: out of memory error\n",(size>>20));while(1);;}
	return ptr;
}
////////////////////////////////////////////////////////////////////////////////
texture<uchar, 3, hipReadModeNormalizedFloat> texRnd;  // 3D texture
texture<uchar, 3, hipReadModeNormalizedFloat> texBrush;  // 3D texture
hipArray *d_rnd_volumeArray = 0;
hipArray *d_brush_volumeArray = 0;
////////////////////////////////////////////////////////////////////////////////
bool init_rnd_texture(int size)
{
	uchar *h_volume = new uchar [size*size*size];

	for(int x = 0; x < size; x++)
	{
	 printf("slice:%d               \r"	,x);
	 for(int y = 0; y < size; y++)
	{for(int z = 0; z < size; z++)
	{
		h_volume[(x)+ (y * size ) + (z * size * size )] = rand()&255;
	}}}

	hipExtent volumeSize = make_hipExtent(size, size, size);

    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    hipMalloc3DArray(&d_rnd_volumeArray, &channelDesc, volumeSize) ;

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_rnd_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams) ;

    // set texture parameters
    texRnd.normalized = true;                      // access with normalized texture coordinates
    texRnd.filterMode = hipFilterModeLinear;      // linear interpolation
    texRnd.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
    texRnd.addressMode[1] = hipAddressModeWrap;
    texRnd.addressMode[2] = hipAddressModeWrap;

    // bind array to 3D texture
    hipBindTextureToArray(texRnd, d_rnd_volumeArray, channelDesc);

	delete []h_volume;

	return true;
}
////////////////////////////////////////////////////////////////////////////////
__device__ int getTerrainVal( 
				  int size_x, int size_y, int size_z, 
				  int x, int y , int z, float scale,
				  float box_x,float box_y,float box_z,
				  float p_x,float p_y,float p_z  )
{
	//p_x = p_y = p_z = 0;
	p_x /= scale;
	p_y /= scale;
	p_z /= scale;
	float add_x = int(p_x+1000);
	float add_y = int(p_y+1000);
	float add_z = int(p_z+1000);
	float frac_x= p_x + 1000 - add_x;
	float frac_y= p_y + 1000 - add_y;
	float frac_z= p_z + 1000 - add_z;
	add_x -= 1000;
	add_y -= 1000;
	add_z -= 1000;

	float x01   = float(x) /  size_x ;
	float y01   = float(y) /  size_y ;
	float z01   = float(z) / (size_z-1) ;

	if (frac_x!=0) if (x01<frac_x) x01+= 1;
	if (frac_y!=0) if (y01<frac_y) y01+= 1;
	if (frac_z!=0) if (z01<frac_z) z01+= 1;

	x01+=add_x;
	y01+=add_y;
	z01+=add_z;

	float xf_in = x01-0.5; xf_in = xf_in*scale;
	float yf_in = y01-0.5; yf_in = yf_in*scale * 0.25;
	float zf_in = z01-0.5; zf_in = zf_in*scale;

	/*
	if ( xf_in > -box_x - 0.5 )
	if ( xf_in < -box_x + 0.5 ) 
	if ( yf_in > -box_y - 0.5 )
	if ( yf_in < -box_y + 0.5 ) 
	if ( zf_in > -box_z - 0.5 )
	if ( zf_in < -box_z + 0.5 ) return -1;
	*/
							   		
	float v=0.0;

	for (float a=0.3,b=0.6;b>0.01;a*=2,b*=0.5)
	{
		v += tex3D(texRnd,0.5+ xf_in*a+a*999, yf_in*a, zf_in*a) * b;
	}
	v += -yf_in*8; 
//	v=0;
	

	if(0)
	for (float objx = -0.1 ; objx <= 0.1 ; objx+=0.02 )
	for (float objz = -0.1 ; objz <= 0.1 ; objz+=0.02 )
	{
		float sinx=sin(objx*327.5);
		float cosx=cos(objx*437.5);
		float sinz=sin(objz*455.9);
		float cosz=cos(objz*655.9);

		float obj_size = 16.0 / 1.0;//tex3D(texRnd, objx * 3.4 , objz * 2.6, 0.35 )*1.0+1.0;

		float xfa = xf_in	+ objx;
		float yfa = yf_in	;//+ 0.25;
		float zfa = zf_in 	+ objz;

		if ( xfa*xfa+yfa*yfa+zfa*zfa > 0.5 ) continue;

		xfa *= obj_size;
		yfa *= obj_size;
		zfa *= obj_size;

		float xfb = xfa * cosx - zfa * sinx;
		float yfb = yfa;
		float zfb = zfa * cosx + xfa * sinx;

		float xf = xfb * sinz - yfb * cosz	;
		float yf = yfb * sinz + xfb * cosz	;
		float zf = zfb						;

//		v = max( v, 1.0-sqrt(xf*xf+yf*yf+zf*zf)*2.0);
		v = max( v, tex3D(texBrush, 2.0*xf+0.5, yf+0.5, zf+0.5) );
//		v +=  tex3D(texBrush, xf+0.5, yf+0.5, zf+0.5) ;
	}					   		
		
	return float( min ( max ( v * 255.0f , 0.0f ), 255.0f )); 
}
////////////////////////////////////////////////////////////////////////////////
__global__ void
cudaTerrainKernel( unsigned int* data, 
				  int size_x , 
				  int size_y , 
				  int size_z , 
				  float anim , float scale,
				  float box_x,float box_y,float box_z,
				  float p_x,float p_y,float p_z  )
{
    extern __shared__ int sdata[];
   
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	int ofs = x+y*(size_x/4);
	int add = size_x*size_y/4;

	for (int z=0;z<size_z-1;z++)
	{
		unsigned int d0 = getTerrainVal( size_x,size_y,size_z, x*4+0, y, z, scale, box_x,box_y,box_z, p_x,p_y,p_z ); //sin(anim*x0)+sin(anim+y0)+sin(anim+z0) ) * (1.0f/6.0f) + 0.5;
		unsigned int d1 = getTerrainVal( size_x,size_y,size_z, x*4+1, y, z, scale, box_x,box_y,box_z, p_x,p_y,p_z ); //tex3D(tex, x1, y0, z0); //( sin(anim*x1)+sin(anim+y0)+sin(anim+z0) ) * (1.0f/6.0f) + 0.5;
		unsigned int d2 = getTerrainVal( size_x,size_y,size_z, x*4+2, y, z, scale, box_x,box_y,box_z, p_x,p_y,p_z ); //tex3D(tex, x2, y0, z0); //( sin(anim*x2)+sin(anim+y0)+sin(anim+z0) ) * (1.0f/6.0f) + 0.5;
		unsigned int d3 = getTerrainVal( size_x,size_y,size_z, x*4+3, y, z, scale, box_x,box_y,box_z, p_x,p_y,p_z ); //tex3D(tex, x3, y0, z0); //( sin(anim*x3)+sin(anim+y0)+sin(anim+z0) ) * (1.0f/6.0f) + 0.5;

		data[ ofs ] = d0 + (d1<<8) + (d2<<16) + (d3 << 24);

		ofs += add;
	}
	data[ ofs ] = data[ x+y*(size_x/4) ];
   
	return;
}
////////////////////////////////////////////////////////////////////////////////
__device__ int getSmoothVal( int size, unsigned int* in_data, int ofs)
{	
	uchar *data = (uchar*) in_data;

	int v = 0;

	for (int x = -1 ; x <= 1 ; x++ )
	for (int y = -1 ; y <= 1 ; y++ )
	for (int z = -1 ; z <= 1 ; z++ )
		v+= data[ofs+x+y*size+z*size*size];

	return v/27;
}
////////////////////////////////////////////////////////////////////////////////
__global__ void
cudaSmoothKernel( unsigned int* out_data, unsigned int* in_data, int size)
{
    extern __shared__ int sdata[];
   
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	int ofs = x+y*(size/4);
	int add = size*size/4;

	if ( x==0 || y==0 || x==size-1 || y==size-1 )
	{
		for (int z=0;z<size;z++)
		{
			out_data[ ofs ] = in_data[ ofs ] ;
			ofs += add;
		}
		return ;
	}

	for (int z=0;z<size;z++)
	{
		if (z>0 && z<size-1 )
		{
			int d0 = getSmoothVal( size, in_data , ofs*4+0);
			int d1 = getSmoothVal( size, in_data , ofs*4+1);
			int d2 = getSmoothVal( size, in_data , ofs*4+2);
			int d3 = getSmoothVal( size, in_data , ofs*4+3);
			out_data[ ofs ] = d0 + (d1<<8) + (d2<<16) + (d3 << 24);
		}
		else
			out_data[ ofs ] = in_data[ ofs ] ;

		ofs += add;
	}
   
	return;
}
////////////////////////////////////////////////////////////////////////////////
__device__ int getBrushVal( int ofs, int size, int x, int y , int z , float anim, unsigned char* data)
{
	float xf_in = 2*float(x) / size - 1.0 ; 
	float yf_in = 2*float(y) / size - 1.0 ; 
	float zf_in = 2*float(z) / size - 1.0 ; 
	float v = 0.0;//max ( 0.0 , 0.05-yf_in );

	//for (float objx = -0.2 ; objx <= 1.2 ; objx+=0.3 )
	//for (float objz = -0.2 ; objz <= 1.2 ; objz+=0.3 )

	float objx = 0.0;
	float objz = 0.0;
	{
		float sinx=sin(objx*7.5);
		float cosx=cos(objx*7.5);
		float sinz=sin(objz*5.9);
		float cosz=cos(objz*5.9);

		float obj_size = 0.7;//tex3D(texRnd, objx * 3.4 , objz * 2.6, 0.35 )*1.0+1.0;

		float xfa = xf_in + objx	+ tex3D(texRnd, objx*4.4 , objz*24.0 , 0.5 )*0.4;
		float yfa = yf_in			- tex3D(texRnd, objx*2.4 , objz*54.0 , 0.5 )*0.9 + 0.8;
		float zfa = zf_in + objz	+ tex3D(texRnd, objx*7.4 , objz*74.0 , 0.5 )*0.4;

		float xfb = xfa * cosx - zfa * sinx;
		float yfb = yfa;
		float zfb = zfa * cosx + xfa * sinx;

		float xf = xfb * sinz - yfb * cosz;
		float yf = yfb * sinz + xfb * cosz;
		float zf = zfb;
						   
		float len = sqrt ( xf*xf + yf*yf + zf*zf );

		//if (len > 0.2/(obj_size*0.3)) continue;

		float nx  = abs(xf) / len;
		float ny  = abs(yf) / len;
		float nz  = abs(zf) / len;

		float az = abs(atan2( xf,yf ));
		float ay = abs(atan2( xf,zf ));
		float ax = abs(atan2( yf,zf ));

		float axz = 0.2;
		float axy = 0.2;
		float ayz = 0.2;
		
		for (float a=0.003,b=0.75;a<1.4;a*=2.0,b*=0.5)
		{
			axz += tex3D(texRnd, objx*3+ax*a,objz*3+az*a		,1.0-objx*3.23	)*b;
			axy += tex3D(texRnd, objx*3+ax*a,objz*3+ay*a		,objz*1.234		)*b;
			ayz += tex3D(texRnd, objx*3+ay*a,objz*3+az*a		,objx+objz*5.22 )*b;
		}
															  
		axz*=tex3D(texRnd, objx*14.4 , objz*15.0 , 0.4 )*2+0.5;
		axy*=tex3D(texRnd, objx*44.4 , objz*74.0 , 0.5 )*2+0.5;
		ayz*=tex3D(texRnd, objx*34.4 , objz*11.0 , 0.7 )*2+0.5;
		

		float displace = 
			max( ayz*(nx*1.0-0.0) , 0.0 ) +
			max( axz*(ny*1.0-0.0) , 0.0 ) +
			max( axy*(nz*1.0-0.0) , 0.0 ) ;
			
	//	v = max( v , 1.0-(1.0-displace*0.8) * obj_size * len );
		float len2 = sqrt ( xf*xf*ayz*ayz + yf*yf*axz*axz + zf*zf*axy*axy );
		v = max( v , 1.0-obj_size * len2 );
		//v = min( v , max(1.0-obj_size * len2,0.0) );
	}

	int vi = float( min ( max ( v * 255.0f , 0.0f ), 255.0f )); 

	return vi;
}
////////////////////////////////////////////////////////////////////////////////
__global__ void
cudaGenBrushKernel( int* data, int size , float anim )
{
    extern __shared__ int sdata[];
   
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	int ofs = x+y*(size/4);
	int add = size*size/4;

	for (int z=0;z<size;z++)
	{
		int d0 = getBrushVal(ofs, size, x*4+0, y, z, anim, (unsigned char *)data); //sin(anim*x0)+sin(anim+y0)+sin(anim+z0) ) * (1.0f/6.0f) + 0.5;
		int d1 = getBrushVal(ofs, size, x*4+1, y, z, anim, (unsigned char *)data); //tex3D(tex, x1, y0, z0); //( sin(anim*x1)+sin(anim+y0)+sin(anim+z0) ) * (1.0f/6.0f) + 0.5;
		int d2 = getBrushVal(ofs, size, x*4+2, y, z, anim, (unsigned char *)data); //tex3D(tex, x2, y0, z0); //( sin(anim*x2)+sin(anim+y0)+sin(anim+z0) ) * (1.0f/6.0f) + 0.5;
		int d3 = getBrushVal(ofs, size, x*4+3, y, z, anim, (unsigned char *)data); //tex3D(tex, x3, y0, z0); //( sin(anim*x3)+sin(anim+y0)+sin(anim+z0) ) * (1.0f/6.0f) + 0.5;

		((unsigned int *)data)[ ofs ] = d0 + (d1<<8) + (d2<<16) + (d3 << 24);

		ofs += add;
	}
   
	return;
}

////////////////////////////////////////////////////////////////////////////////
bool init_brush_texture(int size)
{
	int* h_volume = (int*)gpu_malloc(size*size*size);

	dim3 threads(THREAD_COUNT_X,THREAD_COUNT_Y,1 );
    dim3 grid(	size/(4*threads.x) , size/threads.y ,1 	);

	cudaGenBrushKernel<<< grid, threads, 0>>>	( h_volume , size , 0 );

	hipExtent volumeSize = make_hipExtent(size, size, size);

    // create 3D array
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
    hipMalloc3DArray(&d_brush_volumeArray, &channelDesc, volumeSize) ;

    // copy data to 3D array
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr((void*)h_volume, volumeSize.width*sizeof(uchar), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_brush_volumeArray;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyDeviceToDevice;
    hipMemcpy3D(&copyParams) ;

    // set texture parameters
    texBrush.normalized = true;                      // access with normalized texture coordinates
    texBrush.filterMode = hipFilterModeLinear;      // linear interpolation
    texBrush.addressMode[0] = hipAddressModeClamp;   // wrap texture coordinates
    texBrush.addressMode[1] = hipAddressModeClamp;
    texBrush.addressMode[2] = hipAddressModeClamp;

    // bind array to 3D texture
    hipBindTextureToArray(texBrush, d_brush_volumeArray, channelDesc);

	return true;
}
////////////////////////////////////////////////////////////////////////////////
extern "C" void* cuda_main( int pbo_out, 
						   int size_x, 
						   int size_y, 
						   int size_z, 
						   int level, 
						   float box_x,float box_y,float box_z,
						   float pos_x,float pos_y,float pos_z )
{
	static bool init1		= init_rnd_texture( 128 );
	static bool init2		= init_brush_texture( 128 );
	if(pbo_out==0) return NULL;
	
	int t0 = timeGetTime();

    unsigned int* out_data;   
    CUDA_SAFE_CALL(cudaGLMapBufferObject( (void**)&out_data, pbo_out));   
	if(out_data==0) return NULL;

	dim3 threads(THREAD_COUNT_X,THREAD_COUNT_Y,1 );
    dim3 grid	(size_x/(4*threads.x) , size_y/threads.y ,1 	);

	float anim = float((timeGetTime()>>4)&1023)*2*M_PI / 1024;

	cudaTerrainKernel<<< grid, threads, 0>>> ( 
		out_data, 
		size_x , 
		size_y, 
		size_z , 
		anim, float(int(1<<level)),
		box_x,box_y,box_z,
		pos_x,pos_y,pos_z );
	
	CUT_CHECK_ERROR("cudaRender failed");

	CUDA_SAFE_CALL( hipDeviceSynchronize() );
	int t2 = timeGetTime();

	printf ("cuda terrain time=%d , scale=%d  \n",t2-t0,int(1<<level));
    
   CUDA_SAFE_CALL(cudaGLUnmapBufferObject( pbo_out));

	return 0;
}
////////////////////////////////////////////////////////////////////////////////
bool pboRegister(int pbo)
{
    // register this buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLRegisterBufferObject(pbo));
	CUT_CHECK_ERROR("cudaGLRegisterBufferObject failed");
//	C_CHECK_GL_ERROR();
	return true;
}
////////////////////////////////////////////////////////////////////////////////
void pboUnregister(int pbo)
{
    // unregister this buffer object with CUDA
    CUDA_SAFE_CALL(cudaGLUnregisterBufferObject(pbo));	
	CUT_CHECK_ERROR("cudaGLUnregisterBufferObject failed");
//	C_CHECK_GL_ERROR();
}
////////////////////////////////////////////////////////////////////////////////


	/*
	float v=0;
	for (float a=0.04,b=0.5;a<0.8;a*=2.0,b*=0.5)
	{
		v = v + tex3D(tex, xf*a+a*sin(yf*14)*0.03, yf*a, zf*a+a*cos(yf*14)*0.03) * b * 0.8;
		//float t=tex3D(tex, xf*a+a*sin(yf*14)*0.03, yf*a, zf*a+a*cos(yf*14)*0.03);
		
		v = v + (cos(xf*a*1600)*0.5+cos(yf*a*600)+cos(zf*a*1400)*0.5+3)*b*0.3333*0.5*0.2;

//		v = v + tex3D(tex, xf*a, yf*a, zf*a) * b;
	}
	*/

